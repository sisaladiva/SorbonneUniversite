/**************************************************************
Lokman A. Abbas-Turki code

Those who re-use this code should mention in their code
the name of the author above.
***************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>


// Function that catches the error 
void testCUDA(hipError_t error, const char* file, int line) {

	if (error != hipSuccess) {
		printf("There is an error in file %s at line %d\n", file, line);
		exit(EXIT_FAILURE);
	}
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


/*One-Dimensional Normal Law. Cumulative distribution function. */
double NP(double x) {
	const double p = 0.2316419;
	const double b1 = 0.319381530;
	const double b2 = -0.356563782;
	const double b3 = 1.781477937;
	const double b4 = -1.821255978;
	const double b5 = 1.330274429;
	const double one_over_twopi = 0.39894228;
	double t;

	if (x >= 0.0) {
		t = 1.0 / (1.0 + p * x);
		return (1.0 - one_over_twopi * exp(-x * x / 2.0) * t * (t * (t *
			(t * (t * b5 + b4) + b3) + b2) + b1));
	}
	else {/* x < 0 */
		t = 1.0 / (1.0 - p * x);
		return (one_over_twopi * exp(-x * x / 2.0) * t * (t * (t * (t *
			(t * b5 + b4) + b3) + b2) + b1));
	}
}


__global__ void init_curand_state_k(hiprandState *state)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(0, idx, 0, &state[idx]);
}


__global__ void MC_k(float S_0, float r, float sigma, float dt, float K, 
					int N, hiprandState *state, float *sum, int n){

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprandState localState = state[idx];
	float2 G;
	float S = S_0;
	extern __shared__ float A[];

	float* R1s, * R2s;
	R1s = A;
	R2s = R1s + blockDim.x;

	for (int i = 0; i < N; i++) {
		G = hiprand_normal2(&localState);
		S *= (1+r*dt*dt+sigma*dt*G.x);
	}
	R1s[threadIdx.x] = expf(-r * dt * dt * N) * fmaxf(0.0f, S - K)/ n;
	R2s[threadIdx.x] = R1s[threadIdx.x] * R1s[threadIdx.x] * n;

	__syncthreads();
	int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			R1s[threadIdx.x] += R1s[threadIdx.x + i];
			R2s[threadIdx.x] += R2s[threadIdx.x + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		atomicAdd(sum, R1s[0]);
		atomicAdd(sum + 1, R2s[0]);
	}
	/* Copy state back to global memory */
	//state[idx] = localState;
}

int main(void) {

	int NTPB = 1024;
	int NB = 1024;
	int n = NB * NTPB;
	float T = 1.0f;
	float S_0 = 50.0f;
	float K = S_0;
	float sigma = 0.2f;
	float r = 0.1f;
	int N = 100;
	float dt = sqrtf(T/N);
	float *sum;
	hipMallocManaged(&sum, 2*sizeof(float));
	hipMemset(sum, 0, 2*sizeof(float));


	hiprandState* states;
	hipMalloc(&states, n*sizeof(hiprandState));
	init_curand_state_k<<<NB, NTPB>>>(states);

	float Tim;
	hipEvent_t start, stop;			// GPU timer instructions
	hipEventCreate(&start);			// GPU timer instructions
	hipEventCreate(&stop);				// GPU timer instructions
	hipEventRecord(start, 0);			// GPU timer instructions

	MC_k<<<NB, NTPB, 2*NTPB*sizeof(float)>>>(S_0, r, sigma, dt, K, 
											N, states, sum, n);

	hipEventRecord(stop, 0);			// GPU timer instructions
	hipEventSynchronize(stop);			// GPU timer instructions
	hipEventElapsedTime(&Tim,			// GPU timer instructions
		start, stop);					// GPU timer instructions
	hipEventDestroy(start);			// GPU timer instructions
	hipEventDestroy(stop);				// GPU timer instructions


	printf("The estimated price is equal to %f\n", sum[0]);
	printf("error associated to a confidence interval of 95%% = %f\n",
		1.96 * sqrt((double)(sum[1] - (sum[0] * sum[0])))/sqrt((double)n));
	printf("The true price %f\n", S_0 * NP((r + 0.5 * sigma * sigma)/sigma) -
									K * expf(-r) * NP((r - 0.5 * sigma * sigma) / sigma));
	printf("Execution time %f ms\n", Tim);


	hipFree(sum);
	hipFree(states);

	return 0;
}